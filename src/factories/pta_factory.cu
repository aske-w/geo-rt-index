#include "factories/pta_factory.hpp"
#include "helpers/time.hpp"

using std::make_unique;
using namespace geo_rt_index::factories;
using geo_rt_index::helpers::cuda_buffer;

PointToAABBFactory::PointToAABBFactory(const std::vector<Point>& _points)
	: points_d(std::move(make_unique<cuda_buffer>())),
      aabb_d(std::move(make_unique<cuda_buffer>()))

{
	MEASURE_TIME("Uploading points to GPU",
		points_d->alloc_and_upload(_points);
	);
	num_points = _points.size();
}

void PointToAABBFactory::SetQuery(types::Aabb query)
{
	if (aabb_d->raw_ptr != nullptr)
		aabb_d->free();

	aabb_d->alloc_and_upload<OptixAabb>({query.ToOptixAabb(2, 4)});
}

std::unique_ptr<OptixBuildInput> PointToAABBFactory::Build()
{
	static const constexpr uint32_t flags[] = {
	    OPTIX_GEOMETRY_FLAG_NONE
	};
	auto bi = make_unique<OptixBuildInput>();
	memset(&*bi, 0, sizeof(OptixBuildInput));
	bi->type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;

	auto& prim= bi->customPrimitiveArray;
	prim.aabbBuffers = (hipDeviceptr_t*) &(aabb_d->raw_ptr);
	prim.numPrimitives = 1;
	prim.numSbtRecords = 1;
	prim.strideInBytes = sizeof(OptixAabb);
	prim.flags = flags;
	prim.sbtIndexOffsetBuffer = 0;
	return std::move(bi);
}

geo_rt_index::Point* PointToAABBFactory::GetPointsDevicePointer() const
{
	return reinterpret_cast<Point*>(this->points_d->raw_ptr);
}

size_t PointToAABBFactory::GetNumPoints() const
{
	return this->num_points;
}
