#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "types.hpp"
#include "launch_parameters.hpp"
#include "optix_helpers.cuh"
#include <optix.h>

#include <limits>
#include <cstdint>
#include <hip/hip_runtime.h>

extern "C" __constant__ launch_parameters params;


extern "C" __global__ void __closesthit__test() {
	printf("__closesthit__test\n");
    // do nothing
}


extern "C" __global__ void __miss__test() {
	printf("__miss__test\n");
    // do nothing
}


// this function is called for every potential ray-aabb intersection
extern "C" __global__ void __intersection__test() {
	const uint32_t primitive_id = optixGetPrimitiveIndex();
	printf("__intersection__test %u\n", primitive_id);
//	printf("Is frontface hit: %x ", optixIsFrontFaceHit());
//	printf("Is backface hit: %x ", optixIsBackFaceHit());
//	printf("result_count %u\n", params.result_count);
//	printf("result_d %llX\n", params.result_d);
//	printf("access %u\n", params.result_d[x]);
	auto x = optixGetPayload_0();
	params.result_d[x] = primitive_id;
//	printf("write");
	optixSetPayload_0(x + 1);
//	set_payload_32(primitive_id);
}


// this function is called for every reported (i.e. confirmed) ray-primitive intersection
extern "C" __global__ void __anyhit__test() {
	const uint32_t primitive_id = optixGetPrimitiveIndex();
	printf("__anyhit_test %d\n", primitive_id);
	set_payload_32(primitive_id);
	optixIgnoreIntersection();
}


// this is the entry point
extern "C" __global__ void __raygen__test() {
	printf("__raygen_test\n");
	constexpr const uint32_t ray_flags = 0;
	const constexpr float t_max= 100;
	const float3 origin {0.5,1.5,0.5};
	const float3 direction {t_max,1.5,0.5};
	uint32_t i0 = 0;
	optixTrace(params.traversable, origin, direction, 0, t_max, 0.0f, OptixVisibilityMask(255), ray_flags, 0, 0,
			   0, i0);
	printf("__raygen_test:%d\n",i0);
//	for(uint i = 0; i < 1000; i++)
//	{
//	}
//	for (float i = -1; i < 1.0f; i += 0.1f)
//		for (float j = -1; j < 1.0f; j += 0.1f)
//			for (float k = -1; k < 1.0f; k += 0.1f)
//			{
//				float3 direction {
//				    i,j,k
//				};
//				optixTrace(params.traversable, origin, direction, -10, 10, 100.0f, OptixVisibilityMask(255), ray_flags, 0, 0,
//				           0, i0);
////
//////				params.result_d[count++] = i0;
//			}
}
