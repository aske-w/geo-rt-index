
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

__global__ void VecAdd(int* A,
                       int* B,
                       int* C) {
    int i = threadIdx.x;
    printf("tIdx=%d\n", i);
    C[i] = A[i] + B[i];
}

int main() {
    const constexpr int N = 3;
    const constexpr int size = N * sizeof(int);
    int A[] = {1, 2, 3};
    int B[] = {10, 10, 10};
    int C[] = {0, 0, 0};

    int* d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = N;
    int blocksPerGrid = 1;

    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    std::cout << std::setprecision(2);
    std::cout << C[0] << '\n'
    << C[1] << '\n'
    << C[2] << '\n';
}