//#include "configuration.hpp"
#include "factories/aabb_factory.hpp"
#include "factories/curve_factory.hpp"
#include "factories/factory.hpp"
#include "factories/point_factory.hpp"
#include "factories/pta_factory.hpp"
#include "factories/triangle_input_factory.hpp"
#include "helpers/argparser.hpp"
#include "helpers/data_loader.hpp"
#include "helpers/input_generator.hpp"
#include "helpers/optix_pipeline.hpp"
#include "helpers/optix_wrapper.hpp"
#include "helpers/pretty_printers.hpp"
#include "helpers/time.hpp"
#include "launch_parameters.hpp"
#include "optix_function_table_definition.h"
#include "optix_stubs.h"
#include "types.hpp"

#include <vector>

// #include "device_code.cu"


using std::unique_ptr;
using std::unique_ptr;

using namespace geo_rt_index;
using helpers::optix_pipeline;
using helpers::optix_wrapper;
using helpers::cuda_buffer;
using factories::Factory;
using factories::PointToAABBFactory;

OptixTraversableHandle foo(optix_wrapper& optix, Factory<OptixBuildInput>& inputFactory) {
	OptixTraversableHandle handle{0};
	unique_ptr<OptixBuildInput> bi = inputFactory.Build();

	OptixAccelBuildOptions bo {};
	memset(&bo, 0, sizeof(OptixAccelBuildOptions));
	bo.buildFlags = OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	bo.operation = OPTIX_BUILD_OPERATION_BUILD;
	bo.motionOptions.numKeys = 1;

	OptixAccelBufferSizes structure_buffer_sizes;
	memset(&structure_buffer_sizes, 0, sizeof(OptixAccelBufferSizes));
	OPTIX_CHECK(optixAccelComputeMemoryUsage(optix.optix_context, &bo, &*bi,
	                                         1, // num_build_inputs
	                                         &structure_buffer_sizes))
	auto uncompacted_size = structure_buffer_sizes.outputSizeInBytes;

	cuda_buffer compacted_size_buffer;
	compacted_size_buffer.alloc(sizeof(uint64_t));

	OptixAccelEmitDesc emit_desc;
	emit_desc.type   = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
	emit_desc.result = compacted_size_buffer.cu_ptr();

	// ==================================================================
	// execute build (main stage)
	// ==================================================================
	cuda_buffer uncompacted_structure_buffer;
	uncompacted_structure_buffer.alloc(structure_buffer_sizes.outputSizeInBytes);
	cuda_buffer temp_buffer;
	temp_buffer.alloc(structure_buffer_sizes.tempSizeInBytes);

	OPTIX_CHECK(optixAccelBuild(optix.optix_context, optix.stream, &bo, &*bi, 1, temp_buffer.cu_ptr(),
	                            temp_buffer.size_in_bytes, uncompacted_structure_buffer.cu_ptr(),
	                            uncompacted_structure_buffer.size_in_bytes, &handle, &emit_desc, 1))
	hipDeviceSynchronize();
	CUERR
	temp_buffer.free();
	return handle;
}

int main(const int argc, const char** argv) {
	geo_rt_index::helpers::ArgParser parser{argc, argv};
	auto args = parser.Parse();

    const constexpr bool debug = false;
    optix_wrapper optix(debug);
    optix_pipeline pipeline(&optix);
    hipDeviceSynchronize(); CUERR


    cuda_buffer /*curve_points_d,*/ as;
//	const uint32_t num_points = (1 << 29) + (1 << 28) + (1 << 26); // = 872,415,232 = 7.76 GB worth of points
//	const uint32_t num_points = (1 << 25) + (3 * 1 << 23) + (1 << 22); // = 62,914,560
//	const uint32_t num_in_range = 1 << 23;
	const auto query = types::Aabb{0,0,1,1};
//	const auto space = Aabb{-180, -90, 180, 90};
//	const bool shuffle = !DEBUG;
	std::vector<Point> points;
	MEASURE_TIME("Generating points",
//		points = InputGenerator::Generate(query, space, num_points, num_in_range, shuffle);
	 	points = DataLoader::Load(args.files);
	);
	const auto num_points = points.size();
	const uint32_t num_in_range{4'194'304};
#if INDEX_TYPE == 1
	PointToAABBFactory f{points};
	f.SetQuery(query);

#else
//	TriangleFactory f{};
	AabbFactory f{};
//	PointFactory f{};
#endif

	unique_ptr<cuda_buffer> result_d = std::make_unique<cuda_buffer>();
	auto result = std::make_unique<bool*>(new bool[num_points]);
	memset(*result, 0, num_points);
	result_d->alloc(sizeof(bool) * num_points);
	result_d->upload(*result, num_points);
	uint32_t device_hit_count = 0;
	cuda_buffer hit_count_d;
	hit_count_d.alloc(sizeof(uint32_t));
	hit_count_d.upload(&device_hit_count, 1);

	auto handle = foo(optix, f);
	LaunchParameters launch_params
	{
		.traversable = handle,
#if INDEX_TYPE == 1
		.points = f.GetPointsDevicePointer(),
		.num_points = points.size(),
#endif
		.result_d = result_d->ptr<bool>(),
		.hit_count = hit_count_d.ptr<uint32_t>(),
		.query_aabb = query
	};

	printf("launch parms num_points %u\n", launch_params.num_points);

	cuda_buffer launch_params_d;
	launch_params_d.alloc(sizeof(launch_params));
	launch_params_d.upload(&launch_params, 1);
	hipDeviceSynchronize(); CUERR

	MEASURE_TIME("Optix launch",
		OPTIX_CHECK(optixLaunch(
			pipeline.pipeline,
			optix.stream,
			launch_params_d.cu_ptr(),
			launch_params_d.size_in_bytes,
			&pipeline.sbt,
	#if SINGLE_THREAD
			1
	#else
			num_points,
	#endif
			1,
			1
		))
		hipDeviceSynchronize();
	);
	CUERR
//	std::cout << points.at(912706) << std::endl;
//	std::cout << points.at(1692308) << std::endl;
//	std::cout << points.at(3947100) << std::endl;
//	std::cout << points.at(5000653) << std::endl;
//	std::cout << points.at(8974027) << std::endl;
//	std::cout << points.at(num_points-1) << std::endl;


//	bool res[num_points];
	MEASURE_TIME("result_d->download", result_d->download(*result, num_points););
	MEASURE_TIME("hit_count_d.download", hit_count_d.download(&device_hit_count, 1););
	MEASURE_TIME("Result check",
		uint32_t hit_count = 0;
		for(uint32_t i = 0; i < num_points; i++)
		{
			if ((*result)[i])
			{
				hit_count++;
	//			std::cout << std::to_string(i) << '\n';
			}
		}
		std::cout << std::to_string(hit_count) << '\n';
		std::cout << std::to_string(device_hit_count) << '\n';
		if(args.debug)
	    {
			assert(hit_count == num_in_range);
			assert(device_hit_count == num_in_range);
	    }
	);


	return 0;
}